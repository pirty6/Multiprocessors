#include "hip/hip_runtime.h"
/*----------------------------------------------------------------

* Multiprocesadores: Cuda matrix multiplication

* Fecha: 26-Sep-2015

* Autor: A01205559 Roberto Nuñez

* X = 1_000, Y = 2_000
  Speedup =  1392.76660  / 0.00310 = 535679.462

*--------------------------------------------------------------*/

 #include "hip/hip_runtime.h"
 #include <stdio.h>
 #include <stdlib.h>
 #include <string.h>
 #include "cheader.h"

 #define X 1000
 #define Y 2000
 #define BLOCK_SIZE 1


 __global__ void multiply(int* a, int* b, int* c, int x, int y) {
   int row = blockIdx.y * blockDim.y + threadIdx.y;
   int col = blockIdx.x * blockDim.x + threadIdx.x;

   int temp =  0;
   if(row < x && col < x) {
     for(int i = 0; i < y; i++) {
       temp += a[row * y + i] * b[i * x + col];
     }
   }
   c[row * x + col] = temp;
 }

int main() {
  double acum = 0;
  int* matrixA;
  int* matrixB;
  // int matrixA[X * Y] = {1,-1,1,2,2,3,-2,-3,-1};
  // int matrixB[Y * X] = {1,0,4,0,2,5,1,3,0};
  // int matrixA[X * Y] = {1,2,-3,4,0,-2};
  // int matrixB[Y * X] = {3,1,2,4,-1,5};

  int* matrixC;
  int* d_matrixA, *d_matrixB, *d_matrixC;

  matrixA = (int*)malloc(sizeof(int) * X * Y);
  matrixB = (int*)malloc(sizeof(int) * X * Y);
  matrixC = (int*)malloc(sizeof(int) * X * X);

  for(int i = 0; i < Y * X; i++) {
    matrixA[i] = (i % Y) + 1;
  }

  for(int i = 0; i < Y * X; i++) {
    matrixB[i] = (i % X) + 1;
  }

  printf("Matrix A:\n");
  for(int i = 0; i < X * Y; i++) {
    if(i % Y == 0) printf("\n");
    printf("%i ", matrixA[i]);
  }
  printf("\n");
  printf("Matrix B:\n");
  for(int i = 0; i < Y * X; i++) {
    if(i % X == 0) printf("\n");
    printf("%i ", matrixB[i]);
  }
  printf("\n");

  hipMalloc((void**)&d_matrixA, sizeof(int) * X * Y);
  hipMalloc((void**)&d_matrixB, sizeof(int) * X * Y);
  hipMalloc((void**)&d_matrixC, sizeof(int) * X * X);

  hipMemcpy(d_matrixA, matrixA, sizeof(int) * X * Y, hipMemcpyHostToDevice);
  hipMemcpy(d_matrixB, matrixB, sizeof(int) * X * Y, hipMemcpyHostToDevice);

  unsigned int grid_rows = (X + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int grid_cols = (Y + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 dimGrid(grid_cols, grid_rows);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  for(int i = 0; i < 10; i++) {
    start_timer();
    multiply<<<dimGrid,dimBlock>>>(d_matrixA, d_matrixB, d_matrixC, X, Y);
    acum += stop_timer();
  }

  hipMemcpy(matrixC, d_matrixC, sizeof(int) * X * X, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  printf("Result:\n");
  for(int i = 0; i < X * X; i++) {
    if(i % X == 0) printf("\n");
    printf("%i ", matrixC[i]);
  }
  printf("\n");

  printf("avg time = %.5f ms\n", (acum / 10));
  hipFree(d_matrixA);
  hipFree(d_matrixB);
  hipFree(d_matrixC);

  free(matrixA);
  free(matrixB);
  free(matrixC);

  return 0;
}
