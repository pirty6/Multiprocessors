#include "hip/hip_runtime.h"
/*----------------------------------------------------------------

*

* Multiprocesadores: Cuda

* Fecha: 3-Dic-2019

* Autor: A01205559 Roberto Nuñez

* Image = 1080 x 1920
  Speedup =  104.82820 ms / 0.00520 ms  = 20159.2692

*--------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>
#include <opencv/highgui.h>
#include "hip/hip_runtime.h"
#include "cppheader.h"

__global__ void grayscale(unsigned char *src, unsigned char *dest, int width,
                          int height, int nChannels) {
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if(y < height && x < width) {
    int pos = (y * width + x) * nChannels;

    float r = src[pos + 2];
    float g = src[pos + 1];
    float b = src[pos + 0];

    dest[pos + 2] = ((0.393f * r + 0.769f * g + 0.189f * b) > 255) ? 255 : (0.393f * r + 0.769f * g + 0.189f * b);
    dest[pos + 1] = ((0.349f * r + 0.686f * g + 0.168f * b) > 255) ? 255 : (0.349f * r + 0.686f * g + 0.168f * b);
    dest[pos + 0] = ((0.272f * r + 0.534f * g + 0.131f * b) > 255) ? 255 : (0.272f * r + 0.534f * g + 0.131f * b);
  }
}

int main(int argc, char* argv[]) {
  int i, size;
	double acum;
	unsigned char *dev_src, *dev_dest;
  Timer t;

	if (argc != 2) {
		printf("usage: %s [dir_image_source]\n", argv[0]);
		return -1;
	}

	IplImage *src = cvLoadImage(argv[1], CV_LOAD_IMAGE_COLOR);
	IplImage *dest = cvCreateImage(cvSize(src->width, src->height), IPL_DEPTH_8U, 3);
	if (!src) {
		printf("Could not load image file: %s\n", argv[1]);
		return -1;
	}

	size = src->width * src->height * src->nChannels * sizeof(uchar);
	hipMalloc((void**) &dev_src, size);
	hipMalloc((void**) &dev_dest, size);

	hipMemcpy(dev_src, src->imageData, size, hipMemcpyHostToDevice);

  dim3 dimGrid(ceil((float)src->width / 16),
               ceil((float)src->height / 16));
  dim3 dimBlock(16, 16, 1);

	acum = 0;
	for (i = 0; i < 10; i++) {
		t.start();
		grayscale<<<dimGrid, dimBlock>>>(dev_src, dev_dest, src->width, src->height, src->nChannels);
		acum += t.stop();
	}

	hipMemcpy(dest->imageData, dev_dest, size, hipMemcpyDeviceToHost);

	hipFree(dev_dest);
	hipFree(dev_src);

	printf("avg time = %.5lf ms\n", (acum / 10));

	cvShowImage("Original", src);
	cvShowImage("Sepia", dest);
	cvWaitKey(0);
	cvDestroyWindow("Original");
	cvDestroyWindow("Sepia");

	return 0;
}
