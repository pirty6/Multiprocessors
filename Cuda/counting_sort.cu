#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE 1000000

__global__ void counting_sort(int* array, int *temp, int size) {
  int i, j, count;
  i = threadIdx.x + (blockIdx.x * blockDim.x);
  if (i < size) {
    count = 0;
    for(j = 0; j < size; j++) {
      if(array[j] < array[i]) {
        count++;
      } else if(array[i] == array[j] && j < i) {
        count++;
      }
    }
    temp[count] = array[i];
  }
}

int main(int argc, char* argv[]) {
  int i, *array;
  int *d_array, *d_temp;

  double ms;

  array = (int*) malloc(sizeof(int) * SIZE);
  random_array(array, SIZE);
  display_array("before", array);

  hipMalloc((void**) &d_array, SIZE * sizeof(int));
  hipMalloc((void**) &d_temp, SIZE * sizeof(int));

  hipMemcpy(d_array, array, SIZE * sizeof(int), hipMemcpyHostToDevice);

  printf("Starting...\n");
  for(i = 0; i < 10; i++) {
    start_timer();
    counting_sort<<<SIZE / THREADS, THREADS>>>(d_array, d_temp, SIZE);
    ms += stop_timer();
  }

  hipMemcpy(array, d_temp, SIZE * sizeof(int), hipMemcpyDeviceToHost);
  display_array("after", array);

  printf("avg time = %5ld ms\n", (ms / N));
  hipFree(d_array); hipFree(d_temp);
  free(array);
  return 0;
}
