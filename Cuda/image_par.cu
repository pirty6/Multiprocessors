#include "hip/hip_runtime.h"
/*----------------------------------------------------------------

*

* Multiprocesadores: Cuda

* Fecha: 11-Nov-2019

* Autor: A01206747 Mariana Perez
  Autor: A01205559 Roberto Nuñez

* Image = 1080 x 1920
  Speedup =  33.93700 ms / 0.00250 ms  = 13.5748

*--------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>
#include <opencv/highgui.h>
#include "hip/hip_runtime.h"
#include "cheader.h"

__global__ void grayscale(unsigned char *src, unsigned char *dest, int width,
                          int height, int nChannels) {
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if(y < height && x < width) {
    int pos = (y * width + x) * nChannels;

    unsigned char r = (float)src[pos];
    unsigned char g = (float)src[pos + 1];
    unsigned char b = (float)src[pos + 2];
    dest[pos] = dest[pos + 1] = dest[pos + 2] = (unsigned char)0.2126 * r + 0.7152 * g + 0.0722 * b;
  }
}

int main(int argc, char* argv[]) {
  int i, size;
	double acum;
	unsigned char *dev_src, *dev_dest;
  Timer t;

	if (argc != 2) {
		printf("usage: %s source_file\n", argv[0]);
		return -1;
	}

	IplImage *src = cvLoadImage(argv[1], CV_LOAD_IMAGE_COLOR);
	IplImage *dest = cvCreateImage(cvSize(src->width, src->height), IPL_DEPTH_8U, 3);
	if (!src) {
		printf("Could not load image file: %s\n", argv[1]);
		return -1;
	}

	size = src->width * src->height * src->nChannels * sizeof(uchar);
	hipMalloc((void**) &dev_src, size);
	hipMalloc((void**) &dev_dest, size);

	hipMemcpy(dev_src, src->imageData, size, hipMemcpyHostToDevice);

  dim3 dimGrid(ceil((float)src->width / 16),
               ceil((float)src->height / 16));
  dim3 dimBlock(16, 16, 1);

	acum = 0;
	for (i = 0; i < 10; i++) {
		t.start();
		grayscale<<<dimGrid, dimBlock>>>(dev_src, dev_dest, src->width, src->height, src->nChannels);
		acum += t.stop();
	}

	hipMemcpy(dest->imageData, dev_dest, size, hipMemcpyDeviceToHost);

	hipFree(dev_dest);
	hipFree(dev_src);

	printf("avg time = %.5lf ms\n", (acum / 10));

	cvShowImage("Original", src);
	cvShowImage("Grayscale", dest);
	cvWaitKey(0);
	cvDestroyWindow("Original");
	cvDestroyWindow("Grayscale");
  cvSaveImage("out.jpg", dest);

	return 0;
}
