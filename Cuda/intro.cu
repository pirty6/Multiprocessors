#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ float fx(float a, float b) {
  return a + b;
}

__global__ void kernel(void) {
  printf("res = %f\n", fx(1.0, 2.0));
}

int main(int argc, char* argv[]) {
  kernel <<<1,1>>>();
  hipDeviceSynchronize();
  
  return 0;
}
