#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel(void) {
  printf("GPU bockIdx %i threadIdx %i: Hello World!\n", blockIdx.x, threadIdx.x);
}

int main(int argc, char* argv[]) {
  kernel <<<6,2>>>();
  hipDeviceSynchronize();

  return 0;
}
